/* hello_world.cu
 * ---------------------------------------------------
 * A Hello World example in CUDA
 * ---------------------------------------------------
 * This is a short program which uses multiple CUDA
 * threads to calculate a "Hello World" message which
 * is then printed to the screen.  It's intended to
 * demonstrate the execution of a CUDA kernel.
 * ---------------------------------------------------
 */
#define SIZE 12
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/* CUDA kernel used to calculate hello world message */
__global__ void hello_world(char *a, int N);

int main(int argc, char **argv)
{
   /* data that will live on host */
   char *data;

   /* data that will live in device memory */
   char *d_data;

   /* allocate and initialize data array */
   data = (char*) malloc(SIZE*sizeof(char));
   data[0]  =  72; data[1]  = 100; data[2]  = 106;
   data[3]  = 105; data[4]  = 107; data[5]  =  27;
   data[6]  =  81; data[7]  = 104; data[8]  = 106;
   data[9]  =  99; data[10] =  90; data[11] =  22;

   /* print data before kernel call */
   printf("Contents of data before kernel call: %s\n", data);

   /* allocate memory on device */
   hipMalloc(&d_data, SIZE*sizeof(char));

   /* copy memory to device array */
   hipMemcpy(d_data, data, SIZE, hipMemcpyHostToDevice);

   /* call kernel */
   hello_world<<<4,3>>>(d_data, SIZE);

   /* copy data back to host */
   hipMemcpy(data, d_data, SIZE, hipMemcpyDeviceToHost);

   /* print contents of array */
   printf("Contents of data after kernel call:  %s\n",data);

   /* clean up memory on host and device */
   hipFree(d_data);
   free(data);
   return(0);
}

/* hello_world
 * Each thread increments an element of the input
 * array by its global thread id
 */
__global__ void hello_world(char *a, int N)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   if(i < N) a[i] = a[i] + i;
}

